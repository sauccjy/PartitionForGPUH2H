#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	int Graph_D_H::Graph::partition(const int goalHeight)
	{
		partition_Tree.push_back(pairs(0, NodeNumber));
		ID_hash.assign(NodeNumber, 0);
		TreeHeight = 1;
		thrust::sort(NE_P.begin(), NE_P.end(), myPair_longitude_less_than());
		bool longitudeLast = true;

		int maxSize = NodeNumber;

		long long int lastPartitionTree = 0;
		long long int nowPartitionTree = 1;
		while (TreeHeight < goalHeight)
		{
			long long int tempMax = -1;
			for (long long int i = lastPartitionTree; i < nowPartitionTree; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				int mid = (right - left) / 2 + left;
				if (left == right) { //empty
					partition_Tree.push_back(pairs(left, left));
					partition_Tree.push_back(pairs(left, left));
					continue;
				}
				if (right - left == 1) {
					partition_Tree.push_back(pairs(left, right));
					partition_Tree.push_back(pairs(right, right));
					continue;
				}
				if (right - left == 2) {
					partition_Tree.push_back(pairs(left, right - 1));
					partition_Tree.push_back(pairs(right - 1, right));
					continue;
				}
				//left -> mid; mid+1 ->  right
				if (longitudeLast)
				{
					thrust::sort(NE_P.begin() + left, NE_P.begin() + mid, myPair_latitude_less_than());
					thrust::sort(NE_P.begin() + mid, NE_P.begin() + right, myPair_latitude_less_than());
				}
				else
				{
					thrust::sort(NE_P.begin() + left, NE_P.begin() + mid, myPair_longitude_less_than());
					thrust::sort(NE_P.begin() + mid, NE_P.begin() + right, myPair_longitude_less_than());
				}

				partition_Tree.push_back(pairs(left, mid));
				partition_Tree.push_back(pairs(mid, right));

				tempMax = ((right - mid) > tempMax) ? (right - mid) : tempMax;
			}
			TreeHeight++;
			longitudeLast = !longitudeLast;
			lastPartitionTree = nowPartitionTree;
			nowPartitionTree = partition_Tree.size();
			maxSize = tempMax;
		}

		thrust::host_vector<int> Degree(NodeNumber);
		for (int i = 0; i < NodeNumber; i++)
		{
			Degree[i] = adjList[i].size();
		}

		int lowestIndexStart = std::pow(2, TreeHeight - 1) - 1;
		int lowestIndexEnd = std::pow(2, TreeHeight) - 2;
		for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
		{
			thrust::sort_by_key(thrust::host, Degree.begin() + partition_Tree[i].first, Degree.begin() + partition_Tree[i].second,
				NE_P.begin() + partition_Tree[i].first);
		}
		for (int i = 0; i < NodeNumber; i++)
		{
			ID_hash[NE_P[i].NodeID] = i;
		}
		lowestMaxSize = maxSize;
		return maxSize;
	}

	int Graph::partition_Latitude_first(int goalHeight)
	{
		partition_Tree.push_back(pairs(0, NodeNumber));
		ID_hash.assign(NodeNumber, 0);
		TreeHeight = 1;
		thrust::sort(NE_P.begin(), NE_P.end(), myPair_latitude_less_than());
		bool longitudeLast = true;

		//int maxSize = NE_P.size() / 2 + (int)(NE_P.size() % 2 != 0);
		int maxSize = NodeNumber;

		int lastPartitionTree = 0;
		int nowPartitionTree = 1;
		while (TreeHeight < goalHeight)
		{
			int tempMax = -1;
			for (int i = lastPartitionTree; i < nowPartitionTree; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				int mid = (right - left) / 2 + left;
				if (left == right) { //empty
					partition_Tree.push_back(pairs(left, left));
					partition_Tree.push_back(pairs(left, left));
					continue;
				}
				if (right - left == 1) {
					partition_Tree.push_back(pairs(left, right));
					partition_Tree.push_back(pairs(right, right));
					continue;
				}
				if (right - left == 2) {
					partition_Tree.push_back(pairs(left, right - 1));
					partition_Tree.push_back(pairs(right - 1, right));
					continue;
				}
				//left -> mid; mid+1 ->  right
				if (!longitudeLast)
				{
					thrust::sort(NE_P.begin() + left, NE_P.begin() + mid, myPair_latitude_less_than());
					thrust::sort(NE_P.begin() + mid, NE_P.begin() + right, myPair_latitude_less_than());
				}
				else
				{
					thrust::sort(NE_P.begin() + left, NE_P.begin() + mid, myPair_longitude_less_than());
					thrust::sort(NE_P.begin() + mid, NE_P.begin() + right, myPair_longitude_less_than());
				}

				partition_Tree.push_back(pairs(left, mid));
				partition_Tree.push_back(pairs(mid, right));

				tempMax = ((right - mid) > tempMax) ? (right - mid) : tempMax;
			}
			TreeHeight++;
			longitudeLast = !longitudeLast;
			lastPartitionTree = nowPartitionTree;
			nowPartitionTree = partition_Tree.size();
			maxSize = tempMax;
		}

		//thrust::host_vector<int> Degree(NodeNumber);
		//for (int i = 0; i < NodeNumber; i++)
		//{
		//	Degree[i] = CSR_node_OutdegreePoint[NE_P[i].NodeID + 1] - CSR_node_OutdegreePoint[NE_P[i].NodeID];
		//}

		//int lowestIndexStart = std::pow(2, TreeHeight - 1) - 1;
		//int lowestIndexEnd = std::pow(2, TreeHeight) - 2;
		//for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
		//{
		//	thrust::sort_by_key(thrust::host, Degree.begin() + partition_Tree[i].first, Degree.begin() + partition_Tree[i].second,
		//		NE_P.begin() + partition_Tree[i].first);
		//}
		for (int i = 0; i < NodeNumber; i++)
		{
			ID_hash[NE_P[i].NodeID] = i;
		}
		lowestMaxSize = maxSize;
		return maxSize;
	}



	int  Graph_D_H::Graph::generateBFS(std::map<int, std::map<int, int>>& graph, std::map<int, bool>& visited_s, std::map<int, bool>& visited_t, int s, double beta) {

		if (graph.find(s) == graph.end()) {
			cout << s << " is not in graph" << endl;
			return -1;
		}
		//visited_s[s] = true;
		if (graph.size() < 2)
		{
			cout << s << " is in graph but graph size < 2" << endl;
			return 0;
		}

		//std::cout << "\t In partition graph: (";
		//for (auto& it : graph) {
		//	std::cout << it.first << ",";
		//}
		//cout << ")" << endl;;

		int target_size = (int)(beta * (double)(graph.size()));
		//cout << "target_size: " << target_size << endl;
		std::queue<int> q;
		q.push(s);
		//visited_s[s] = true;
		int markSize = 0;
		//cout << "\t BFS chosen ID: (";
		//cout << s;

		while (!q.empty()) {
			int nodeNow = q.front();
			q.pop();
			if (visited_t[nodeNow]|| visited_s[nodeNow]) {
				continue;
			}
			visited_s[nodeNow] = true;
			//cout << nodeNow << ",";
			markSize++;
			if (markSize > target_size)
				break;
			for (auto& it : graph[nodeNow]) {
				if (!visited_s[it.first]) {
					q.push(it.first);
				}
			}
		}

		//while (!q.empty() && bfs_result.size() < target_size) {
		//	int level_size = q.size();
		//	std::vector<int> current_level;

		//	for (int i = 0; i < level_size; ++i) {
		//		int u = q.front();
		//		q.pop();
		//		bfs_result.push_back(u);
		//		current_level.push_back(u);

		//		for (const auto& v : graph[u]) {
		//			if (!visited_s[v.first]) {
		//				visited_s[v.first] = true;
		//				q.push(v.first);
		//			}
		//		}
		//	}

		//	// Sort current level by degree (number of neighbors)
		//	std::sort(current_level.begin(), current_level.end(), [&graph](int a, int b) {
		//		return graph[a].size() < graph[b].size();
		//		});

		//	// Update BFS result with sorted current level
		//	for (const int& node : current_level) {
		//		if (bfs_result.size() >= target_size) break;
		//		bfs_result.push_back(node);
		//	}
		//}

		//// Mark the nodes in bfs_result as visited
		//for (const int& node : bfs_result) {
		//	visited_s[node] = true;
		//	cout << node<<",";
		//}
		//cout << ")\n";

		return markSize;
	}


	void Graph_D_H::Graph::generateFarthestVertex(std::map<int, std::map<int, int>>& standard_graph, int s, int& f) {
		queue<pair<int, int>> run_que;
		run_que.push(make_pair(s, 1));
		vector<bool> visited_s(NodeNumber, false);
		int tempFarthest = s;
		//visited_s[s] = true;
		int tempMaxHeight = 1;
		//cout << "searching node: " << s << "'th farthest node" << endl;
		while (!run_que.empty()) {
			auto& it = run_que.front();
			int tempID = it.first;
			int tempHeight = it.second;
			run_que.pop();

			if (visited_s[tempID]) continue;

			visited_s[tempID] = true;
			if (tempHeight > tempMaxHeight) {
				tempFarthest = tempID;
				tempMaxHeight = tempHeight;
			}
			else {
				if (standard_graph[tempID].size() < standard_graph[tempFarthest].size() ) {
					tempFarthest = tempID;
					tempMaxHeight = tempHeight;
				}
			}
			//cout << "height: " << tempHeight << " is: " << tempID << endl;
			for (auto& it : standard_graph[tempID]) {
				if (!visited_s[it.first]) {
					run_que.push(make_pair(it.first, tempHeight + 1));
					
				}
					//run_que.push(make_pair(it.first, tempHeight + 1));
			}

		}
		f = tempFarthest;
	}


	bool Graph_D_H::Graph::bfs_for_edmondsKarp(const std::map<int, std::map<int, int>>& residualGraph, int source, int sink, std::map<int, int>& parent) {
		std::map<int, bool> visited;
		std::queue<int> q;
		q.push(source);
		visited[source] = true;
		parent[source] = -1;

		while (!q.empty()) {
			int u = q.front();
			q.pop();

			if (residualGraph.find(u) == residualGraph.end()) continue; // Check if u exists in residualGraph

			for (const auto& v : residualGraph.at(u)) {
				if (!visited[v.first] && v.second > 0) {
					q.push(v.first);
					parent[v.first] = u;
					visited[v.first] = true;
					if (v.first == sink)
						return true;
				}
			}
		}

		return false;
	}

	// Function to implement the Edmonds-Karp algorithm
	int  Graph_D_H::Graph::edmondsKarp(std::map<int, std::map<int, int>>& graph, int source, int sink, std::map<int, std::map<int, int>>& residualGraph) {
		residualGraph = graph; // Initialize residual graph
		std::map<int, int> parent;
		int maxFlow = 0;

		// Augment the flow while there is a path from source to sink
		while (bfs_for_edmondsKarp(residualGraph, source, sink, parent)) {
			int pathFlow = INT_MAX;

			// Find the maximum flow through the path found by BFS
			for (int v = sink; v != source; v = parent[v]) {
				int u = parent[v];
				pathFlow = std::min(pathFlow, residualGraph[u][v]);
			}

			// Update residual capacities of the edges and reverse edges along the path
			for (int v = sink; v != source; v = parent[v]) {
				int u = parent[v];
				residualGraph[u][v] -= pathFlow;
				residualGraph[v][u] += pathFlow;
			}

			// Add path flow to the overall flow
			maxFlow += pathFlow;
		}

		return maxFlow;
	}

	// Function to find the minimum cut using the residual graph
	void  Graph_D_H::Graph::findMinCut(const std::map<int, std::map<int, int>>& residualGraph, int source, std::map<int, bool>& visited) {
		std::queue<int> q;
		q.push(source);
		visited[source] = true;

		while (!q.empty()) {
			int u = q.front();
			q.pop();

			if (residualGraph.find(u) == residualGraph.end()) continue; // Check if u exists in residualGraph

			for (const auto& v : residualGraph.at(u)) {
				if (!visited[v.first] && v.second > 0) {
					q.push(v.first);
					visited[v.first] = true;
				}
			}
		}
	}

	// Main function to find the minimum cut and split the graph
	void  Graph_D_H::Graph::findMinCutAndSplitGraph(std::map<int, std::map<int, int>>& graph, int source, int sink, 
		std::set<int>& source_paritition, std::set<int>& sink_paritition) {
		// Convert undirected graph to directed graph with unit capacity

		std::map<int, std::map<int, int>> residualGraph;
		int maxFlow = edmondsKarp(graph, source, sink, residualGraph);
		//std::cout << "Maximum flow: " << maxFlow << std::endl;

		std::map<int, bool> visited;
		for (const auto& u : residualGraph) {
			visited[u.first] = false;
		}
		findMinCut(residualGraph, source, visited);

		for (const auto& u : graph) {
			if (visited.at(u.first)) {
				source_paritition.insert(u.first);
			}
			else {
				sink_paritition.insert(u.first);
			}
		}
	}






	int Graph_D_H::Graph::partition_minumum_cut(int goalHeight, double beta) {
		PartitionMethod = 2;
		this->Beta = beta;
		partition_Tree.push_back(pairs(0, NodeNumber));
		ID_hash.assign(NodeNumber, 0);
		TreeHeight = 1;
		thrust::host_vector<myPair<double> > NE_P_copy = NE_P;
		int maxSize = NodeNumber;

		int lastPartitionTree = 0;
		int nowPartitionTree = 1;
		while (TreeHeight < goalHeight)
		{
			int tempMax = -1;

			cout << "_________________at layer:" << TreeHeight << "____________" << endl;
			for (int i = lastPartitionTree; i < nowPartitionTree; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				//cout << "in partition: <" << left << "." << right << "," << endl;
				if (left == right) { //empty
					partition_Tree.push_back(pairs(left,left));
					partition_Tree.push_back(pairs(left, left));
					continue;
				}
				if (right - left == 1) {
					partition_Tree.push_back(pairs(left, right));
					partition_Tree.push_back(pairs(right, right));
					continue;
				}
				if (right - left == 2) {
					partition_Tree.push_back(pairs(left, right - 1));
					partition_Tree.push_back(pairs(right - 1, right));
					continue;
				}

				//create copy and delete edges which not in this partition
				std::map<int, std::map<int, int>> standard_graph_in_partition;
				map<int, bool> firstPartition, secondPartition;
				int randomNode = -1;
				for (int j = left; j < right; j++) {

					int NodeID = NE_P[j].NodeID;
					randomNode = NodeID;

					standard_graph_in_partition.emplace(NodeID, std::map<int, int>());
					firstPartition.emplace(NodeID, false);
					secondPartition.emplace(NodeID, false);
				}
				
				for (int j = left; j < right; j++) {

					int NodeID = NE_P[j].NodeID;
					randomNode = (adjList[NodeID].size() > adjList[randomNode].size()) ? NodeID : randomNode;
					for (auto& it : adjList[NodeID]) {
						if (standard_graph_in_partition.find(it.first) != standard_graph_in_partition.end())
							standard_graph_in_partition[NodeID].emplace(it.first, 1);
					}
				}
				//displayGraph(standard_graph_in_partition);
				//generate two farthest vertex
				
				int s = -1, t = -1;
				generateFarthestVertex(standard_graph_in_partition, randomNode, s);
				generateFarthestVertex(standard_graph_in_partition, s, t);
				secondPartition.at(t) = true;
				generateBFS(standard_graph_in_partition, firstPartition, secondPartition, s, beta);
				secondPartition.at(t) = false;
				generateBFS(standard_graph_in_partition, secondPartition,firstPartition, t, beta);
				//std::cout << "choose id: " << s << " and " << t << endl;

				//delete such patititon( if exists v in both S and T, put them in mid partition , if not in S and T , push them too)
				standard_graph_in_partition.emplace(NodeNumber + 1, std::map<int, int>());//S
				standard_graph_in_partition.emplace(NodeNumber + 2, std::map<int, int>());//T
				std::set<int> s_partition = {}, t_partition = {}, mid_partiiton = {}, merge_partition = {};
				for (auto& io : firstPartition) {
					int ID = io.first;
					if (io.second && !secondPartition[ID]) {
						s_partition.insert(ID);
					}
					else if (!io.second && secondPartition[ID]) {
						t_partition.insert(ID);
					}
					//else if (!io.second && !secondPartition[ID]) {
					//	mid_partiiton.insert(ID);
					//}
					else {
						mid_partiiton.insert(ID);
						//merge_partition.insert(ID);
					}
				}
				//if (TreeHeight == 1) {
				//	cout << "s size: " << s_partition.size() << " t size: " << t_partition.size() <<" mid size: "<<mid_partiiton.size()
				//		<<" merge size: "<< merge_partition.size()<< endl;
				//}
				//if (s_partition.size() > t_partition.size()) {
				//	t_partition.insert(merge_partition.begin(), merge_partition.end());
				//}
				//else {
				//	s_partition.insert(merge_partition.begin(), merge_partition.end());
				//}

				//std::cout << "vertex in s_partition: ";
				//for (auto& it : s_partition) {
				//	cout << it << ",";
				//}
				//cout << endl;
				//std::cout << "vertex in t_partition: ";
				//for (auto& it : t_partition) {
				//	cout << it << ",";
				//}
				//cout << endl;
				//std::cout << "vertex in mid_partition: ";
				//for (auto& it : mid_partiiton) {
				//	cout << it << ",";
				//}
				//cout << endl;

				queue<int> answer_queue = {};
				map<int, bool> is_visited = {};
				for (auto& io : firstPartition) {
					is_visited.emplace(io.first, false);
				}
				answer_queue.push(s);
				while (!answer_queue.empty()) {
					int nodenow = answer_queue.front();
					answer_queue.pop();
					if (is_visited.find(nodenow)->second) {
						continue;
					}
					is_visited[nodenow] = true;
					for (auto& it : standard_graph_in_partition[nodenow]) {
						if (s_partition.find(it.first) != s_partition.end()) {
							if (!is_visited.find(it.first)->second)
								answer_queue.push(it.first);
							continue;
						}
						else {
							standard_graph_in_partition[it.first].emplace(NodeNumber + 1, 1);
							standard_graph_in_partition[it.first].erase(nodenow);
							standard_graph_in_partition[NodeNumber + 1].emplace(it.first, 1);
						}
					}
				}

				for (auto& its : s_partition) {
					standard_graph_in_partition[its].clear();
					standard_graph_in_partition.erase(its);
				}

				is_visited.clear();
				for (auto& io : firstPartition) {
					is_visited.emplace(io.first, false);
				}
				answer_queue.push(t);
				while (!answer_queue.empty()) {
					int nodenow = answer_queue.front();
					answer_queue.pop();
					if (is_visited.find(nodenow)->second) {
						continue;
					}
					is_visited[nodenow] = true;
					for (auto& it : standard_graph_in_partition[nodenow]) {
						if (t_partition.find(it.first) != t_partition.end()) {
							if (!is_visited.find(it.first)->second)
								answer_queue.push(it.first);
							continue;
						}
						else {
							standard_graph_in_partition[it.first].emplace(NodeNumber + 2, 1);
							standard_graph_in_partition[it.first].erase(nodenow);
							standard_graph_in_partition[NodeNumber + 2].emplace(it.first, 1);
						}
					}
				}

				for (auto& its : t_partition) {
					standard_graph_in_partition[its].clear();
					standard_graph_in_partition.erase(its);
				}
				//displayGraph(standard_graph_in_partition);
				//find minimum cut
				std::set<int> source_paritition = {};
				std::set<int> sink_paritition = {};
				findMinCutAndSplitGraph(standard_graph_in_partition, NodeNumber + 1, NodeNumber + 2, source_paritition, sink_paritition);

				int sourceSizeGap = s_partition.size() + source_paritition.size() - (t_partition.size() + sink_paritition.size());
				
				std::set<int> source_paritition_2 = {};
				std::set<int> sink_paritition_2 = {};
				findMinCutAndSplitGraph(standard_graph_in_partition, NodeNumber + 2, NodeNumber + 1, source_paritition_2, sink_paritition_2);
				int sourceSizeGap2 = s_partition.size() + source_paritition_2.size() - (t_partition.size() + sink_paritition_2.size());

				if (sourceSizeGap > sourceSizeGap2) {
					source_paritition.clear();
					source_paritition.insert(source_paritition_2.begin(), source_paritition_2.end());
					source_paritition_2.clear();
					sink_paritition.clear();
					sink_paritition.insert(sink_paritition_2.begin(), sink_paritition_2.end());
					sink_paritition_2.clear();
				}

				if (source_paritition.find(NodeNumber + 1) != source_paritition.end()) {
					source_paritition.erase(NodeNumber + 1);
					source_paritition.insert(s_partition.begin(), s_partition.end());
					sink_paritition.erase(NodeNumber + 2);
					sink_paritition.insert(t_partition.begin(), t_partition.end());
				}
				else {
					source_paritition.erase(NodeNumber + 2);
					source_paritition.insert(t_partition.begin(), t_partition.end());
					sink_paritition.erase(NodeNumber + 1);
					sink_paritition.insert(s_partition.begin(), s_partition.end());
				}

				int mid = source_paritition.size() + left;

				vector<myPair<double> > NE_P_first = {};
				//thrust::host_vector<myPair<double> > NE_P_second = {};
				for (auto& it : source_paritition) {
					NE_P_first.push_back(myPair<double>(NE_P_copy[it].first, NE_P_copy[it].second, it));
				}
				for (auto& it : sink_paritition) {
					NE_P_first.push_back(myPair<double>(NE_P_copy[it].first, NE_P_copy[it].second, it));
				}
				if (NE_P_first.size() == right - left) {
					thrust::copy(NE_P_first.begin(), NE_P_first.end(), NE_P.begin() + left);
				}

				partition_Tree.push_back(pairs(left, mid));
				partition_Tree.push_back(pairs(mid, right));

				tempMax = ((right - mid) > tempMax) ? (right - mid) : tempMax;

			}
			TreeHeight++;



			lastPartitionTree = nowPartitionTree;
			nowPartitionTree = partition_Tree.size();
			maxSize = tempMax;
		}


		//thrust::host_vector<int> Degree(NodeNumber);
		//for (int i = 0; i < NodeNumber; i++)
		//{
		//	Degree[i] = CSR_node_OutdegreePoint[NE_P[i].NodeID + 1] - CSR_node_OutdegreePoint[NE_P[i].NodeID];
		//}

		//int lowestIndexStart = std::pow(2, TreeHeight - 1) - 1;
		//int lowestIndexEnd = std::pow(2, TreeHeight) - 2;
		//for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
		//{
		//	thrust::sort_by_key(thrust::host, Degree.begin() + partition_Tree[i].first, Degree.begin() + partition_Tree[i].second,
		//		NE_P.begin() + partition_Tree[i].first);
		//}
		for (int i = 0; i < NodeNumber; i++)
		{
			ID_hash[NE_P[i].NodeID] = i;
		}
		lowestMaxSize = maxSize;
		return maxSize;
	}



	void Graph::makeAdjcentNode_2(int goalHeight)
	{
		Graph_D_H::time_Mine time;

		cout << "start construct Partition Rank tree " << endl;
		time.updateStart();

		int tempHeight = TreeHeight;
		thrust::host_vector<bool> isAdjcent(NodeNumber, true);
		//thrust::host_vector<pairs> range(NodeNumber, pairs(INT_MAX, INT_MIN));
		thrust::host_vector<int> placeOccupy(NodeNumber, 0);

		for (int i = 0; i < NodeNumber; i++) {
			placeOccupy[i] = adjList[i].size();
		}

		NANHash.assign(partition_Tree.size(), myPair<int>());
		nonAdjcentNode.clear();
		vector<double> afs(TreeHeight + 1, 0);
		while (tempHeight > goalHeight)
		{
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			thrust::host_vector<bool> isAdjcent_cp = isAdjcent;
			for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
			{
				NANHash[i].first = nonAdjcentNode.size();//index
				NANHash[i].second = 0; //maxSize
				NANHash[i].NodeID = 0;//nonAdjcent Size
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				//vector<int> outPartitionSize(right - left, 0);
				map<int, int> subgraph;
				for (int j = left; j < right; j++)//construct sub graph
				{
					if (!isAdjcent[NE_P[j].NodeID])
						continue;
					int nodeID = NE_P[j].NodeID;
					subgraph.emplace(nodeID, 0);
				}
				NANHash[i].second = subgraph.size();//record max size
				for (auto& it : subgraph) {//define outsize
					it.second = subgraph.size();
					for (auto& adj : adjList[it.first]) {
						if (!isAdjcent[adj.first]) {
							continue;
						}
						if (subgraph.find(adj.first) == subgraph.end()) {
							it.second++;
							//isAdjcent_cp[it.first] = false;
						}
					}
				}

				for (auto& it : subgraph) {
					if (it.second == NANHash[i].second) {// mark non-adjNode
						isAdjcent_cp[it.first] = false;
						NANHash[i].NodeID++;
						nonAdjcentNode.push_back(TDrank(adjList[it.first].size(), it.first, 0));
					}
					placeOccupy[it.first] = max(placeOccupy[it.first], it.second);

				}
			}
			isAdjcent = isAdjcent_cp;
			afs[tempHeight] = nonAdjcentNode.size();
			tempHeight--;
		}

		time.updateEnd();
		//makePartitionRankTreeTime = time.get_microsecond_duration();
		cout << "\t construct Partition Rank tree end,  using time: " << time.get_microsecond_duration() << endl;
		string latitudePartition = "LatitudePartitionRate.csv";
		if (PartitionMethod == 2) latitudePartition = "MinimumPartitionRate.csv";
		std::fstream heightdata(latitudePartition, ios::in | ios::out | ios::app);

		heightdata << graphName << ",";
		cout << "NodeNumber per layer: \n";
		for (int i = 1; i < afs.size(); i++) {
			heightdata << afs[i] / NodeNumber << ",";
			cout << "\t layer : " << i << " size: " << afs[i] << endl;
		}
		heightdata << "\n";
		heightdata.close();


		Reflash_for_PRT.assign(nonAdjcentNode.size(), -1);
		for (auto i = 0; i < nonAdjcentNode.size(); i++) {
			int ID = nonAdjcentNode[i].second;
			Reflash_for_PRT[ID] = i;
		}
		//Reflash_for_PRT_D = Reflash_for_PRT;

		//allowcate CHTree
		cout << "start Allocate NUB " << endl;
		time.updateStart();
		CHTree.clear();
		CHTreeHash.assign(NodeNumber, myPair<int>());

		for (int i = 0; i < NodeNumber; i++)
		{
			int nodeID = NE_P[i].NodeID;
			CHTreeHash[i].first = 0;//actual size
			CHTreeHash[i].second = placeOccupy[nodeID];//maxSize
			//cout << placeOccupy[i] << endl;
			CHTreeHash[i].NodeID = CHTree.size();//CHTree index
			CHTree.insert(CHTree.end(), CHTreeHash[i].second, myPair<int>(INT_MAX, 0, -1)); //insert maxSize label(outpoint,weight,hub)

			for (auto& it : adjList[nodeID])
			{
				CHTree[CHTreeHash[i].NodeID + CHTreeHash[i].first].setPairs(-1, it.first, it.second);
				CHTreeHash[i].first++;
				if (CHTreeHash[i].first > CHTreeHash[i].second) {
					cout << "NUB error~! ::: NodeID: " << nodeID << " " << CHTreeHash[i].first - CHTreeHash[i].second << endl;
					int nonAdjcentNodeIndex = Reflash_for_PRT[nodeID];
					int inHeight = 1;
					while (1) {
						if (afs[inHeight] < nonAdjcentNodeIndex) {
							break;
						}
						inHeight++;
						//afs[inHeight] > 
					}
					cout << "nonAdjcentNodeIndex : " << nonAdjcentNodeIndex << endl;
					cout << "\t actual adjcent size: " << adjList[nodeID].size() << " nub size��" << CHTreeHash[i].second << " at height:" << inHeight - 1 << endl;
					//throw("LUB allocate Error!!");
				}
			}
		}

		chSize = CHTree.size();

		time.updateEnd();
		//AllocateLUBTime = time.get_microsecond_duration();
		cout << "\t Malloc LUB end,  using time: " << time.get_microsecond_duration() << endl;
		//cout << "LUB Size: " << ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024) << "MB" << endl;
		//LUBSize = ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024);
		cout << "LUB Size: " << ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024) << "MB" << endl;

	}


	void Graph_D_H::Graph::displayPartition()
	{
		cout << "______________________PARTITION________________________" << endl;
		int tempHeight = 1;
		while (tempHeight <= TreeHeight)
		{
			cout << "height:" << tempHeight << ";  ";
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			tempHeight++;
			for (int i = lowestIndexStart; i <= lowestIndexEnd; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				if (left == right) {
					cout << "()" ;
					continue;
				}
					
				cout << "(";
				for (int j = left; j < right ; j++)
				{
					cout << NE_P[j].NodeID << ",";
				}
				cout  << ")";
			}
			cout << endl;
		}
		cout << "______________________ID_HASH________________________" << endl;
		for (int i = 0; i < NodeNumber; i++)
		{
			cout << "ID:" << i << " hash:" << ID_hash[i] << "; ";
		}
		cout << endl;
	}

	void  Graph_D_H::Graph::writePartition() {

		std::string fileName;
		if (PartitionMethod == 1) {
			fileName = "./Partition/" + graphName + "/LatitudeCut.txt";
		}
		else if (PartitionMethod == 2) {
			fileName = "./Partition/" + graphName + "/MinimumCut.txt";
		}
		cout << fileName << endl;
		int tempHeight = 1;
		std::fstream heightdata;
		heightdata.open(fileName,  ios::out );
		heightdata << TreeHeight << endl;
		while (tempHeight <= TreeHeight)
		{
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			
			for (int i = lowestIndexStart; i <= lowestIndexEnd; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				heightdata << tempHeight << " " << left << " " << right << "\n";
			}
			
			tempHeight++;
		}

		heightdata.close();


		if (PartitionMethod == 1) {
			fileName = "./Partition/" + graphName +  "/LatitudeID.txt";
		}
		else if (PartitionMethod == 2) {
			fileName = "./Partition/" + graphName + "/MinimumID.txt";
		}
		cout << fileName << endl;
		std::fstream heightdata_ID;
		heightdata_ID.open(fileName, ios::out);
		heightdata_ID << NodeNumber << endl;
		for (auto i = 0; i < NE_P.size(); i++) {
			heightdata_ID << i << " " << std::fixed << std::setprecision(1) <<  (double)NE_P[i].first << " " << (double)NE_P[i].second << " " << NE_P[i].NodeID << endl;
		}
		heightdata_ID.close();

	}

}